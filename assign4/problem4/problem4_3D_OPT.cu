#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <sys/time.h>

#define THRESHOLD (std::numeric_limits<float>::epsilon())

using std::cerr;
using std::cout;
using std::endl;

#define cudaCheckError(ans)                                                    \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}

const uint64_t N = (1 << 7); // Matrix width
const uint64_t SIZE_IN_BYTES_MATRIX = N * N * N * sizeof(float);
#define M 5 // Convolution filter width
#define TILE_WIDTH 4 // Output tile width
#define BLOCK_WIDTH (TILE_WIDTH + M - 1) // Block width

__host__ __device__ bool is_valid_3D(const int i, const int j, const int k,  const uint64_t N){
	return 0 <= i && i < N && 0 <= j && j < N && 0 <= k && k < N;
}

__constant__ float kernel_mat[M][M][M];

__global__ void kernel3D(const float *input, float *output){
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int tz = threadIdx.z;

	int dep_o = blockIdx.z * TILE_WIDTH + tz; // Output depth
	int row_o = blockIdx.y * TILE_WIDTH + ty; // Output row
	int col_o = blockIdx.x * TILE_WIDTH + tx; // Outut column
	
	int dep_i = dep_o - M / 2; // Input depth
	int row_i = row_o - M / 2; // Input row
	int col_i = col_o - M / 2; // Input column

	__shared__ float temp[BLOCK_WIDTH][BLOCK_WIDTH][BLOCK_WIDTH];

	if(is_valid_3D(dep_i, row_i, col_i, N)){
		temp[tz][ty][tx] = input[dep_i * N * N + row_i * N + col_i];
	}
	else{
		temp[tz][ty][tx] = 0;
	}
	__syncthreads();

	if(is_valid_3D(tz, ty, tx, TILE_WIDTH) && is_valid_3D(dep_o, row_o, col_o, N)){
		float sum = 0;
		for(int i = 0; i < M; i++){
			for(int j = 0; j < M; j++){
				for(int k = 0; k < M; k++){
					sum += temp[tz + i][ty + j][tx + k] * kernel_mat[i][j][k];
				}
			}
		}
		output[dep_o * N * N + row_o * N + col_o] = sum / (M * M * M);
	}
}

__host__ void check_result_3D(const float* w_ref, const float* w_opt) {
  double maxdiff = 0.0;
  int numdiffs = 0;

  for (uint64_t i = 0; i < N; i++) {
    for (uint64_t j = 0; j < N; j++) {
      for (uint64_t k = 0; k < N; k++) {
        double this_diff =
            w_ref[i * N * N + j * N + k] - w_opt[i * N * N + j * N + k];
        if (std::fabs(this_diff) > THRESHOLD) {
          numdiffs++;
          if (this_diff > maxdiff) {
            maxdiff = this_diff;
          }
        }
      }
    }
  }

  if (numdiffs > 0) {
    cout << numdiffs << " Diffs found over THRESHOLD " << THRESHOLD
         << "; Max Diff = " << maxdiff << endl;
  } else {
    cout << "No differences found between base and test versions\n";
  }
}

void print3D(const float* A) {
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < N; ++j) {
      for (int k = 0; k < N; ++k) {
        cout << A[i * N * N + j * N + k] << "\t";
      }
      cout << "\n";
    }
    cout << "\n";
  }
}

double rtclock() { // Seconds
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) {
    cout << "Error return from gettimeofday: " << stat << "\n";
  }
  return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void init_matrix_3D(float *a,  const uint64_t N){
		for(int i = 0; i < N; i++){
				for(int j = 0; j < N; j++){
						for(int k = 0; k < N; k++){
								a[i * N * N + j * N + k] = rand() % 100;
						}
				}
		}
}

void calculate_ref_3D(const float *a, float *b, const float *filter){
	float sum;
	for(int i = 0; i < N; i++){
			for(int j = 0; j < N; j++){
				for(int k = 0; k < N; k++){
					sum = 0;		
					for(int di = -M / 2; di <= M / 2; di++){
						for(int dj = -M / 2; dj <= M / 2; dj++){
							for(int dk = -M / 2; dk <= M / 2; dk++){
								if(!is_valid_3D(i + di, j + dj, k + dk, N)){
										continue;
								}
								sum += a[(i + di) * N * N + (j + dj) * N + (k + dk)] * filter[(M / 2 + di) * M * M + (M / 2 + dj) * M + (M / 2 + dk)];
							}
						}
					}
					b[i * N * N + j * N + k] = sum / (M * M * M); 
				}
			}
	}
}

void convolution_3D(){
	int SIZE_IN_BYTES_FILTER = M * M * M * sizeof(float);

	float *a = NULL, *b_ref = NULL, *b = NULL, *filter = NULL;
	a = (float*)malloc(SIZE_IN_BYTES_MATRIX);
	b_ref = (float*)malloc(SIZE_IN_BYTES_MATRIX);
	b = (float*)malloc(SIZE_IN_BYTES_MATRIX);
	filter = (float*)malloc(SIZE_IN_BYTES_FILTER);

	init_matrix_3D(a, N);
	init_matrix_3D(filter, M);

	double clkbegin = rtclock();
	calculate_ref_3D(a, b_ref, filter);
	double clkend = rtclock();
	double cpu_time = clkend - clkbegin;
	cout << "Convolution 3D time on CPU: " << cpu_time * 1000 << " msec " << endl;

  	float *input = NULL, *output = NULL;
	cudaCheckError(hipMalloc(&input, SIZE_IN_BYTES_MATRIX));
	cudaCheckError(hipMalloc(&output, SIZE_IN_BYTES_MATRIX));

	cudaCheckError(hipMemcpy(input, a, SIZE_IN_BYTES_MATRIX, hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(kernel_mat), filter, SIZE_IN_BYTES_FILTER));

	dim3 threadsPerBlock(BLOCK_WIDTH, BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 numBlocks(N / TILE_WIDTH, N/ TILE_WIDTH, N / TILE_WIDTH);

	hipEvent_t start, end;
	hipEventCreate(&start);
	hipEventCreate(&end);
	hipEventRecord(start, 0);
	kernel3D<<<numBlocks, threadsPerBlock>>>(input, output);
	cudaCheckError(hipPeekAtLastError());
	hipEventRecord(end, 0);
	cudaCheckError(hipMemcpy(b, output, SIZE_IN_BYTES_MATRIX, hipMemcpyDeviceToHost));	
	float kernel_time;
	hipEventElapsedTime(&kernel_time, start, end);
	hipEventDestroy(start);
	hipEventDestroy(end);
	check_result_3D(b_ref, b);
	cout << "Convolution 3D time on GPU: " << kernel_time << " msec " << endl;
	
	free(a);
	free(b);
	free(filter);

	hipFree(input);
	hipFree(output);
}

int main() {
  	srand(time(NULL));

	int SIZE_IN_BYTES_FILTER = M * M * M * sizeof(float);

	float *a = NULL, *b_ref = NULL, *b = NULL, *filter = NULL;
	a = (float*)malloc(SIZE_IN_BYTES_MATRIX);
	b_ref = (float*)malloc(SIZE_IN_BYTES_MATRIX);
	b = (float*)malloc(SIZE_IN_BYTES_MATRIX);
	filter = (float*)malloc(SIZE_IN_BYTES_FILTER);

	init_matrix_3D(a, N);
	init_matrix_3D(filter, M);

	double clkbegin = rtclock();
	calculate_ref_3D(a, b_ref, filter);
	double clkend = rtclock();
	double cpu_time = clkend - clkbegin;
	cout << "Convolution 3D time on CPU: " << cpu_time * 1000 << " msec " << endl;

  	float *input = NULL, *output = NULL;
	cudaCheckError(hipMalloc(&input, SIZE_IN_BYTES_MATRIX));
	cudaCheckError(hipMalloc(&output, SIZE_IN_BYTES_MATRIX));

	hipEvent_t start1,start2, end1,end2;
	float kernel_time1,kernel_time2;
	hipEventCreate(&start1);
    hipEventCreate(&start2);
    hipEventCreate(&end1);
    hipEventCreate(&end2);

	dim3 threadsPerBlock(BLOCK_WIDTH, BLOCK_WIDTH, BLOCK_WIDTH);
	dim3 numBlocks(N / TILE_WIDTH, N/ TILE_WIDTH, N / TILE_WIDTH);

	cudaCheckError(hipEventRecord(start1, 0));
	cudaCheckError(hipMemcpy(input, a, SIZE_IN_BYTES_MATRIX, hipMemcpyHostToDevice));
	cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(kernel_mat), filter, SIZE_IN_BYTES_FILTER));

	cudaCheckError(hipEventRecord(start2, 0));
	kernel3D<<<numBlocks, threadsPerBlock>>>(input, output);
	cudaCheckError(hipEventRecord(end2, 0));
	cudaCheckError(hipDeviceSynchronize());
	cudaCheckError(hipMemcpy(b, output, SIZE_IN_BYTES_MATRIX, hipMemcpyDeviceToHost));	
	cudaCheckError(hipEventRecord(end1, 0));
	cudaCheckError(hipDeviceSynchronize());

	cudaCheckError(hipGetLastError());

	check_result_3D(b_ref, b);
	hipEventElapsedTime(&kernel_time1, start1, end1);
    hipEventElapsedTime(&kernel_time2, start2, end2);

	cout << "Convolution 3D time on GPU kernel_time1 for cpu compare: " << kernel_time1 << " msec " << endl;
	cout << "Convolution 3D time on GPU kernel_time2 for gpu compare: " << kernel_time2 << " msec " << endl;
	hipEventDestroy(start1);
	hipEventDestroy(start2);
	hipEventDestroy(end1);
	hipEventDestroy(end2);



	free(a);
	free(b);
	free(filter);

	hipFree(input);
	hipFree(output);
	hipFree(kernel_mat);

  	return EXIT_SUCCESS;
}
