#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <numeric>
#include <sys/time.h>

#define THRESHOLD (std::numeric_limits<double>::epsilon())

using std::cerr;
using std::cout;
using std::endl;

#define cudaCheckError(ans)               \
{                                       \
	gpuAssert((ans), __FILE__, __LINE__); \
}

inline void gpuAssert(hipError_t code, const char *file, int line,
											bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
						line);
		if (abort)
			exit(code);
	}
}

const uint64_t N = (256);

__global__ void kernel1(const double* input, double* output) 
{
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t j = blockIdx.y * blockDim.y + threadIdx.y;
    uint64_t k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i>0 && j>0 && k>0 && i<N-1 && j< N-1 && k < N-1)
    {
        output[i + j * N + k * N * N] =  0.8 * (input[(i - 1) + j * N + k * N * N] + input[(i + 1) + j * N + k* N * N] + input[i + (j - 1) * N + k * N * N] + input[i + (j + 1) * N + k* N * N] + input[i + j * N + (k - 1)* N * N] + input[i + j * N + (k + 1)* N * N]);
    }
}

__host__ void stencil(double* input, double* output) {
    for (uint64_t i=1; i<N-1; i++) 
    {
        for (uint64_t j=1; j<N-1; j++) 
        {
            for (uint64_t k=1; k<N-1; k++) 
            {
                output[i * N * N + j * N + k] = 0.8 * (input[(i - 1) * N * N + j * N + k] + input[(i + 1) * N * N + j * N + k] + input[i * N * N + (j - 1) * N + k] + input[i * N * N + (j + 1) * N + k] + input[i * N * N + j * N + k - 1] + input[i * N * N + j * N + k + 1]);
            }
        }
    }
}

__host__ void check_result(const double *w_ref, const double *w_opt,
													 const uint64_t size)
{
	double maxdiff = 0.0;
	int numdiffs = 0;

	for (uint64_t i = 0; i < size; i++)
	{
		for (uint64_t j = 0; j < size; j++)
		{
			for (uint64_t k = 0; k < size; k++)
			{
				double this_diff =
						w_ref[i + N * j + N * N * k] - w_opt[i + N * j + N * N * k];
				if (std::fabs(this_diff) > THRESHOLD)
				{
					numdiffs++;
					if (this_diff > maxdiff)
					{
						maxdiff = this_diff;
					}
				}
			}
		}
	}

	if (numdiffs > 0)
	{
		cout << numdiffs << " Diffs found over THRESHOLD " << THRESHOLD
				 << "; Max Diff = " << maxdiff << endl;
	}
	else
	{
		cout << "No differences found between base and test versions\n";
	}
}

void print_mat(const double *A)
{
	for (uint64_t i = 0; i < N; ++i)
	{
		for (uint64_t j = 0; j < N; ++j)
		{
			for (uint64_t k = 0; k < N; ++k)
			{
				printf("%lf,", A[i * N * N + j * N + k]);
			}
			printf("      ");
		}
		printf("\n");
	}
}

double rtclock()
{ // Seconds
	struct timezone Tzp;
	struct timeval Tp;
	int stat;
	stat = gettimeofday(&Tp, &Tzp);
	if (stat != 0)
	{
		cout << "Error return from gettimeofday: " << stat << "\n";
	}
	return (Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

int main()
{
	uint64_t SIZE = N * N * N;
	uint64_t SIZE_BYTES = SIZE * sizeof(double);

    double *h_in, *h_out, *h_out_1;
    double *k1_in, *k1_out;

    h_in = new double[SIZE]; 
    h_out = new double[SIZE](); 
    h_out_1 = new double[SIZE]; 

    for(uint64_t i=0;i<N;i++)
    {
        for(uint64_t j=0;j<N;j++)
        {
            for(uint64_t k=0;k<N;k++)
            {
                h_in[i * N * N + j * N + k] = (i - j + k) * 0.1;
            }
        }
    }

    cudaCheckError(hipMalloc(&k1_in, SIZE_BYTES));
    cudaCheckError(hipMalloc(&k1_out, SIZE_BYTES));


	double clkbegin = rtclock();
	stencil(h_in, h_out);
	double clkend = rtclock();

	double cpu_time = clkend - clkbegin;
	cout << "Stencil time on CPU: " << cpu_time * 1000 << " msec" << endl;

	float t1, t2;
    hipEvent_t start1,start2, end1, end2 ;
    hipEventCreate(&start1);
    hipEventCreate(&start2);
    hipEventCreate(&end1);
    hipEventCreate(&end2);

    dim3 k1_TBP(8,8,8);
    dim3 k1_grid (N/8, N/8, N/8);

    cudaCheckError(hipEventRecord(start1, 0));
    cudaCheckError(hipMemcpy(k1_in, h_in, SIZE_BYTES, hipMemcpyHostToDevice));
    cudaCheckError(hipEventRecord(start2, 0));
    kernel1<<<k1_grid, k1_TBP>>>(k1_in, k1_out);
    cudaCheckError(hipEventRecord(end2, 0));
    cudaCheckError(hipEventSynchronize(end2));
    cudaCheckError(hipMemcpy(h_out_1, k1_out, SIZE_BYTES, hipMemcpyDeviceToHost)); 
    cudaCheckError(hipGetLastError());
    cudaCheckError(hipEventRecord(end1, 0));
    cudaCheckError(hipEventSynchronize(end1));

    check_result(h_out, h_out_1, N);

	hipEventElapsedTime(&t2, start2, end2);
	hipEventElapsedTime(&t1, start1, end1);

	std::cout << "Kernel 1 time (ms) (without 2-way memcpy): " << t2 << "\n";
	std::cout << "Kernel 1 time (ms) (with 2-way memcpy): " << t1 << "\n";

    hipFree(k1_in);
    hipFree(k1_out);

    free(h_in);
    free(h_out);
    free(h_out_1);

	return EXIT_SUCCESS;
}
